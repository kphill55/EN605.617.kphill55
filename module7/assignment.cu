#include <vector>
#include <iostream>
#include "benchmarking.h"
#include "assignment.h"

using u32 = unsigned int;

void run_4_kernels(u32 * results, const u32 * const data1, const u32 * const data2,
const u32 n_blocks, const u32 block_size, hipStream_t & stream) {
    add_matrices<<<n_blocks, block_size, 0, stream>>>(results, data1, data2);
    subtract_matrices<<<n_blocks, block_size, 0, stream>>>(results, data1, data2);
    multiply_matrices<<<n_blocks, block_size, 0, stream>>>(results, data1, data2);
    modulo_matrices<<<n_blocks, block_size, 0, stream>>>(results, data1, data2);
	hipStreamSynchronize(stream);
}

void run_kernels(u32 * results, const u32 * const data1, const u32 * const data2,
const u32 n_blocks, const u32 block_size, const size_t array_size) {
	// Create cuda stream
	hipStream_t stream1; 
  	hipStreamCreate(&stream1);

	// Create events
	hipEvent_t start, stop;
	hipEventCreate(&start); 
  	hipEventCreate(&stop);
    float elapsedTime;

    // Allocate device memory
    u32 * device_results;
    hipHostAlloc((void **)&device_results, array_size * sizeof(u32), hipHostMallocDefault);
    u32 * arr1;
    hipHostAlloc((void **)&arr1, array_size * sizeof(u32), hipHostMallocDefault);
    u32 * arr2;
    hipHostAlloc((void **)&arr2, array_size * sizeof(u32), hipHostMallocDefault);

	// Start event
	hipEventRecord(start);

    // Copy data1 memory to GPU memory
    hipMemcpyAsync(arr1, data1,
        array_size * sizeof(u32), hipMemcpyHostToDevice);
    hipMemcpyAsync(arr2, data2,
        array_size * sizeof(u32), hipMemcpyHostToDevice);

    run_4_kernels(device_results, arr1, arr2, n_blocks, block_size, stream1);

    // Flush a message through std out while the GPU works
    std::cout << "I am printing this while the GPU does work..." << std::endl;

	hipMemcpyAsync(results, device_results,
        array_size * sizeof(u32), hipMemcpyDeviceToHost);

	hipStreamSynchronize(stream1); // Wait until the stream is clear
  	hipEventRecord(stop, stream1); // Record the stop event
  	hipEventSynchronize(stop); // Wait until work recorded during the event is done
    hipEventElapsedTime(&elapsedTime, start, stop);

    std::cout << "Event took " << elapsedTime << " ms\n";

    hipFree(arr1);
    hipFree(arr2);
    hipFree(device_results);
	hipStreamDestroy(stream1);
}

int main(int argc, char * argv[]) {
    // Parse command line
    unsigned int block_size = 0; // Threads per block
	unsigned int n_threads = 0; // Total threads we want
	unsigned int n_blocks = 0; // Number of blocks to hold all the threads
	unsigned int N_INTS = 0;
    
    if (argc == 4) {
        n_threads = std::stol(std::string(argv[1]));
		block_size = std::stol(std::string(argv[2]));
		n_blocks = (n_threads / block_size) > 0 ? n_threads / block_size : 1;
		N_INTS = std::stol(std::string(argv[3]));
    }
    else {
        std::cout << "Usage: " << argv[0] << " [block size] [number of threads per block] [array size" << std::endl;
		return 0;
    }

    // Allocate host memory ones vectors for 2 separate runs
    std::vector<u32> ones(N_INTS, 1);
    std::vector<u32> twos(N_INTS, 2);

    // Allocate two host destination vectors
    std::vector<u32> dest(N_INTS);

    // Run shared memory 4 kernels
    TIC();
    run_kernels(dest.data(), ones.data(), twos.data(), n_blocks, block_size, N_INTS);
    std::cout << "Stream/event kernel took " << TOC<std::chrono::microseconds>() << " microseconds" << std::endl;
    print_vector(dest);
}
