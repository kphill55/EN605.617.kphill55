#include <vector>
#include <iostream>
#include "benchmarking.h"
#include "assignment.h"
#include <hipfft/hipfft.h>
#include <hipblas.h>

using c32 = std::complex<float>;
using c64 = std::complex<double>;
using u32 = unsigned int;

void run_fft(c32 * results, const u32 array_num_elements) {
	// Create FFT
    hipfftHandle plan;

    // Set FFT plan
    hipfftPlan1d(&plan, array_num_elements, HIPFFT_C2C, 1);

    
    // Create cuda stream
	hipStream_t stream1;
    hipStreamCreate(&stream1);

    // Set FFT to stream1
    hipfftSetStream(plan, stream1);

	// Create events
	hipEvent_t start, stop;
	hipEventCreate(&start); 
  	hipEventCreate(&stop);
    float elapsedTime = -1;

    // Allocate device memory for FFT
    hipfftComplex * device_results;
    hipMallocManaged((void **)&device_results, array_num_elements * sizeof(c32), hipMemAttachGlobal);

	// Start event
	hipEventRecord(start, stream1);
    TIC();

    // Copy data1 memory to GPU memory
    hipMemcpyAsync(device_results, results,
        array_num_elements * sizeof(c32), hipMemcpyHostToDevice);

    hipfftExecC2C(plan, device_results, device_results, HIPFFT_FORWARD);

    // Flush a message through std out while the GPU works
    std::cout << "I, the CPU, am printing this while the GPU does an FFT..." << std::endl;

	hipMemcpyAsync(results, device_results,
        array_num_elements * sizeof(c32), hipMemcpyDeviceToHost);

    // Push the stop event onto the kernel launch queue after copying the data out
    hipEventRecord(stop, stream1);

    /*
    We must now synchronize with steam1 since this is asynchronous
    unlike stream0 which is synchronous by default
    (host waits for kernel execution to end)
    */
	hipStreamSynchronize(stream1); // Wait for stream1 to complete queued actions
  	hipEventSynchronize(stop); // Wait for "stop" to reach the front of the kernel queue
    
    std::cout << "CPU waited " << TOC<std::chrono::microseconds>() << " microseconds" << std::endl;
    
    // This should be roughly the same as tic/toc
    hipEventElapsedTime(&elapsedTime, start, stop);
    std::cout << "Event took " << elapsedTime*1000 << " microseconds in total\n";

    // Cleanup
    hipFree(device_results);
	hipStreamDestroy(stream1);
    hipfftDestroy(plan);
}

void run_matrix_mult(c32 * results, const c32 * const data1, const c32 * const data2, const u32 x_dim, const u32 y_dim) {
    hipblasHandle_t handle;

    // Create cuda stream
	hipStream_t stream1;
    hipStreamCreate(&stream1);

    // Create blas
    hipblasCreate(&handle);
    hipblasSetStream(handle, stream1);

	// Create events
	hipEvent_t start, stop;
	hipEventCreate(&start); 
  	hipEventCreate(&stop);
    float elapsedTime = -1;

    // Allocate
    hipComplex * device_results;
    hipMallocManaged((void **)&device_results, x_dim * y_dim * sizeof(c32), hipMemAttachGlobal);
    hipComplex * arr1;
    hipMallocManaged((void **)&arr1, x_dim * y_dim * sizeof(c32), hipMemAttachGlobal);
    hipComplex * arr2;
    hipMallocManaged((void **)&arr2, x_dim * y_dim * sizeof(c32), hipMemAttachGlobal);

	// Start event
	hipEventRecord(start, stream1);
    TIC();
    
    // Copy the host matrix to device
    hipblasSetMatrixAsync(x_dim, y_dim, sizeof(c32), data1, x_dim, arr1, y_dim, stream1);
    hipblasSetMatrixAsync(x_dim, y_dim, sizeof(c32), data2, x_dim, arr2, y_dim, stream1);
    
    // Execute
    hipComplex alpha{1,0};
    hipComplex beta{1,0};
    cublasCgemm3m(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, x_dim, y_dim, x_dim, &alpha, arr1, x_dim, arr2, y_dim, &beta, device_results, x_dim);

    // Flush a message through std out while the GPU works
    std::cout << "I, the CPU, am printing this while the GPU does a matrix multiplication..." << std::endl;

    // Retrieve the device matrix back to the host
	hipblasGetMatrixAsync(x_dim, y_dim, sizeof(c32), device_results, x_dim, results, y_dim, stream1);

    // Push the stop event onto the kernel launch queue after copying the data out
    hipEventRecord(stop, stream1);

    /*
    We must now synchronize with steam1 since this is asynchronous
    unlike stream0 which is synchronous by default
    (host waits for kernel execution to end)
    */
	hipStreamSynchronize(stream1); // Wait for stream1 to complete queued actions
  	hipEventSynchronize(stop); // Wait for "stop" to reach the front of the kernel queue
    
    std::cout << "CPU waited " << TOC<std::chrono::microseconds>() << " microseconds" << std::endl;
    
    // This should be roughly the same as tic/toc
    hipEventElapsedTime(&elapsedTime, start, stop);
    std::cout << "Event took " << elapsedTime*1000 << " microseconds in total\n";


    // Clean up
    hipblasDestroy(handle);
	hipStreamDestroy(stream1);
    hipFree(device_results);
    hipFree(arr1);
    hipFree(arr2);
}

int main(int argc, char * argv[]) {
    // Parse command line
	unsigned int N_COMPLEX = 0;
    
    if (argc == 2) {
		N_COMPLEX = std::stol(std::string(argv[1]));
    }
    else {
        std::cout << "Usage: " << argv[0] << " [block size] [number of threads per block] [array size" << std::endl;
		return 0;
    }

    // "Receive" a buffer of complex samples of an "unknown signal"
    float initial_phase = 0.0f;
	float angular_frequency = PI;
	std::vector<c32> rx(N_COMPLEX);

	for (int i = 0; i < rx.size(); ++i) {
		rx[i] = std::polar(1.0f, i * angular_frequency + initial_phase);
	}

	// Create 2d array identity matrix
	std::vector<std::vector<c32>> i_matrix(N_COMPLEX, std::vector<c32> (N_COMPLEX, 0));
	
	for (int i = 0; i<N_COMPLEX; ++i) {
		i_matrix[i][i] = c32{1.0f, 0.0f};
	}
	
	// Create 2d array of twos
	std::vector<std::vector<c32>> twos(N_COMPLEX, std::vector<c32> (N_COMPLEX, {0, 2}));

    std::vector<std::vector<c32>> results(N_COMPLEX, std::vector<c32> (N_COMPLEX));

    // Run FFT
    TIC();
    run_fft(rx.data(), N_COMPLEX);
    std::cout << "Stream/event kernel took " << TOC<std::chrono::microseconds>() << " microseconds" << std::endl;
    print_complex_vector(rx);

    // Run matrix mult
    TIC();
    run_matrix_mult(results.begin()->data(), twos.begin()->data(), i_matrix.begin()->data(), N_COMPLEX, N_COMPLEX);
    std::cout << "Stream/event kernel took " << TOC<std::chrono::microseconds>() << " microseconds" << std::endl;
    print_complex_vector(results.front());    
}
